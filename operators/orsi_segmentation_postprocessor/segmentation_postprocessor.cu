#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "segmentation_postprocessor.cuh"

namespace holoscan::ops::orsi {
namespace segmentation_postprocessor {

__forceinline__ __device__ uint32_t hwc_to_index(Shape shape, uint32_t y, uint32_t x, uint32_t c) {
  return (y * shape.width + x) * shape.channels + c;
}

__forceinline__ __device__ uint32_t nchw_to_index(Shape shape, uint32_t y, uint32_t x, uint32_t c) {
  return (c * shape.height + y) * shape.width + x;
}

template <enum DataFormat>
__forceinline__ __device__ uint32_t data_format_to_index(Shape shape, uint32_t y, uint32_t x,
                                                         uint32_t c) {}

template <>
__forceinline__ __device__ uint32_t data_format_to_index<DataFormat::kHWC>(Shape shape, uint32_t y,
                                                                           uint32_t x, uint32_t c) {
  return hwc_to_index(shape, y, x, c);
}

template <>
__forceinline__ __device__ uint32_t data_format_to_index<DataFormat::kNHWC>(Shape shape, uint32_t y,
                                                                            uint32_t x,
                                                                            uint32_t c) {
  return hwc_to_index(shape, y, x, c);
}

template <>
__forceinline__ __device__ uint32_t data_format_to_index<DataFormat::kNCHW>(Shape shape, uint32_t y,
                                                                            uint32_t x,
                                                                            uint32_t c) {
  return nchw_to_index(shape, y, x, c);
}

__forceinline__ __device__ uint32_t hw1_to_index(Shape shape, uint32_t y, uint32_t x) {
  return y * shape.width + x;
}

template <enum NetworkOutputType network_output_type, enum DataFormat data_format>
__global__ void postprocessing_kernel(Shape shape, const float* input, output_type_t* output) {
  const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= shape.width) || (y >= shape.height)) { return; }

  uint8_t max_index = 0;

  switch (network_output_type) {
    case NetworkOutputType::kSigmoid: {
      const float value = input[data_format_to_index<data_format>(shape, y, x, 0)];
      max_index = value >= 0.5f ? 1 : 0;
    } break;
    case NetworkOutputType::kSoftmax: {
      float max_value = 0.0f;
      for (uint32_t c = 0; c < shape.channels; c++) {
        const float value = input[data_format_to_index<data_format>(shape, y, x, c)];
        if (value > max_value) {
          max_value = value;
          max_index = c;
        }
      }
    } break;
  }

  output[hw1_to_index(shape, y, x)] = max_index;
}

uint16_t ceil_div(uint16_t numerator, uint16_t denominator) {
  uint32_t accumulator = numerator + denominator - 1;
  return accumulator / denominator;
}

template <enum DataFormat data_format>
__global__ void threshold_kernel(Shape shape, output_type_t* input, output_type_t* output) {
  const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  const uint32_t n = shape.width*shape.height;
  if (((x >= shape.width) || (y >= shape.height)) && n>(blockDim.x*blockDim.y)) { return; }

  
  const float value = input[data_format_to_index<data_format>(shape, y, x, 0)];
  output[hw1_to_index(shape, y, x)]  = value >= 0.5f ? 1 : 0;
}

__forceinline__ __device__ double sigmoid (double a)
{
    return 1.0 / (1.0 + exp(-a));
}

template <enum DataFormat data_format>
__global__ void sigmoid_kernel (Shape shape, const float * __restrict__ src, 
                                output_type_t * __restrict__ dst)
{

  const float value = src[0];
  
  dst[0] = (sigmoid(value)>= 0.5f) ? 1 : 0;
}

void cuda_postprocess(enum NetworkOutputType network_output_type, enum DataFormat data_format,
                      Shape shape, const float* input, output_type_t* output,
                      hipStream_t hip_stream) {
  dim3 block(32, 32, 1);
  dim3 grid(ceil_div(shape.width, block.x), ceil_div(shape.height, block.y), 1);
  switch (network_output_type) {
    case NetworkOutputType::kSigmoid:
      switch (data_format) {
        case DataFormat::kNCHW:
          postprocessing_kernel<NetworkOutputType::kSigmoid, DataFormat::kNCHW>
              <<<grid, block, 0, hip_stream>>>(shape, input, output);
          break;
        case DataFormat::kHWC:
          postprocessing_kernel<NetworkOutputType::kSigmoid, DataFormat::kHWC>
              <<<grid, block, 0, hip_stream>>>(shape, input, output);
          break;
        case DataFormat::kNHWC:
          postprocessing_kernel<NetworkOutputType::kSigmoid, DataFormat::kNHWC>
              <<<grid, block, 0, hip_stream>>>(shape, input, output);
          break;
      }
      break;
    case NetworkOutputType::kSoftmax:
      switch (data_format) {
        case DataFormat::kNCHW:
          postprocessing_kernel<NetworkOutputType::kSoftmax, DataFormat::kNCHW>
              <<<grid, block, 0, hip_stream>>>(shape, input, output);
          break;
        case DataFormat::kHWC:
          postprocessing_kernel<NetworkOutputType::kSoftmax, DataFormat::kHWC>
              <<<grid, block, 0, hip_stream>>>(shape, input, output);
          break;
        case DataFormat::kNHWC:
          postprocessing_kernel<NetworkOutputType::kSoftmax, DataFormat::kNHWC>
              <<<grid, block, 0, hip_stream>>>(shape, input, output);
          break;
      }
      break;
      case NetworkOutputType::kRawValues:
      switch (data_format) {
        case DataFormat::kNCHW:
          sigmoid_kernel<DataFormat::kNCHW><<<grid, block>>>(shape, input, output);
          break;
        case DataFormat::kHWC:
          sigmoid_kernel<DataFormat::kHWC>
                <<<grid, block>>>(shape, input, output);
          break;
        case DataFormat::kNHWC:
          sigmoid_kernel<DataFormat::kNHWC>
                <<<grid, block>>>(shape, input, output);
          break;
      }
      break;
  }
}


__global__ void resizing_kernel(Shape input_shape, Shape output_shape, const uint8_t* input, uint8_t* output, int32_t offset_x, int32_t offset_y) {
  const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= input_shape.width) || (y >= input_shape.height)) { return; }

  output[hw1_to_index(output_shape, y + offset_y, x  + offset_x)] = input[hw1_to_index(input_shape, y, x)];
}

void cuda_resize(Shape input_shape, Shape output_shape, const uint8_t* input, uint8_t* output, int32_t offset_x, int32_t offset_y) {
  dim3 block(32, 32, 1);
  dim3 grid(ceil_div(input_shape.width, block.x), ceil_div(input_shape.height, block.y), 1);
  
  resizing_kernel<<<grid, block>>>(input_shape, output_shape, input, output, offset_x, offset_y);
}

}  // namespace segmentation_postprocessor
}  // namespace holoscan::ops
