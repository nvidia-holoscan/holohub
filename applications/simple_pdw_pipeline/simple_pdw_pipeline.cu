#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <arpa/inet.h>
#include <cuda/std/complex>
#include "basic_network_operator_rx.h"
#include "basic_network_operator_tx.h"
#include "holoscan/holoscan.hpp"
#include "matx.h"

using namespace matx;
using ftype = float;
using ComplexType = cuda::std::complex<ftype>;

// Time or frequency series data with a tag for an id
struct TaggedSignalData {
  TaggedSignalData(int _id, tensor_t<ComplexType, 1> _signal) : signal(_signal), id(_id) {}
  int id;
  tensor_t<ComplexType, 1> signal;
};

// A detected pulse.
//
// This represents a pulse that has been found by the threshold detector.
struct DetectedPulseSlice {
  DetectedPulseSlice(int _id, int _offset, int _zero_bin, tensor_t<ComplexType, 1> _signal,
                     tensor_t<ftype, 1> _power)
      : signal(_signal), power(_power), zero_bin(_zero_bin), offset(_offset), id(_id) {}
  // ID the pulse is derived from
  int id;
  // Starting bin of the following tensors.
  int offset;
  // Center bin of the original full signal.
  int zero_bin;
  // Power of the pulse
  tensor_t<ftype, 1> power;
  // Raw signal of the pulse.
  // This is not used in this implementation,
  // But may be used to do advanced pulse charatarization.
  tensor_t<ComplexType, 1> signal;
};

// Processed pulse description
struct PulseDescription {
  PulseDescription(int _id, int _low, int _high, int _zero, float _max_amplitude, float _sum_power,
                   float _average_amplitude)
      : id(_id),
        low_bin(_low),
        high_bin(_high),
        zero_bin(_zero),
        max_amplitude(_max_amplitude),
        sum_power(_sum_power),
        average_amplitude(_average_amplitude) {}
  // Id the pulse was derived from.
  int id;
  // Starting bin of the pulse
  int low_bin;
  // Final bin of the pulse
  int high_bin;
  // Center of the pulse in the original full signal.
  int zero_bin;
  // Maximum amplitude of pulse
  float max_amplitude;
  // Total power of pulse
  float sum_power;
  // Average amplitude of pulse
  float average_amplitude;
};

// Number of int32s in the network report message.
const unsigned int burst_length = 7;

// Displays or prints the pulse descriptions.
class PulsePrinterOp : public holoscan::Operator {
 public:
  HOLOSCAN_OPERATOR_FORWARD_ARGS(PulsePrinterOp);
  PulsePrinterOp() = default;

  void setup(holoscan::OperatorSpec& spec) override {
    spec.input<PulseDescription>("pulse_description");
    spec.output<NetworkOpBurstParams>("burst_out");
    // Sample rate to display the Hz on screen.
    spec.param(sample_rate, "sample_rate", "Samples per second", "Sample rate in Hz.", {});
    // True if this Operator prints to the screen.
    spec.param(to_screen, "to_screen", "Print data to screen", "Print Data to screen", {});
    // True if this Operator transmits on the udp network.
    spec.param(to_tx, "to_tx", "Send Data through Tx", "Send data through Tx", {});
  }

  void compute(holoscan::InputContext& op_input, holoscan::OutputContext& op_output,
               holoscan::ExecutionContext&) override {
    auto data = op_input.receive<PulseDescription>("pulse_description");
    // Frequency resolution of the FFT
    float resolution_mhz = (sample_rate.get() / (2e6 * data->zero_bin));
    float low_freq = (data->low_bin - data->zero_bin) * resolution_mhz;
    float high_freq = (data->high_bin - data->zero_bin) * resolution_mhz;
    // Print to screen
    if (to_screen.get()) {
      std::cout << "Sum power " << data->sum_power << std::endl;
      std::cout << "Pulse started at " << low_freq << " MHz" << std::endl;
      std::cout << "Pulse end at " << high_freq << " MHz" << std::endl;
      std::cout << "Max power:" << data->max_amplitude << std::endl;
      std::cout << "Average amplitude " << data->average_amplitude << std::endl;
    }

    auto buff = new uint32_t[7];
    // Next, place these values into a network burst operator and send it
    if (to_tx.get()) {
      buff[0] = htonl((uint32_t)data->id);
      buff[1] = htonl((uint32_t)data->low_bin);
      buff[2] = htonl((uint32_t)data->high_bin);
      buff[3] = htonl((uint32_t)data->zero_bin);
      buff[4] = htonl((uint32_t)data->sum_power);
      buff[5] = htonl((uint32_t)data->max_amplitude);
      buff[6] = htonl((uint32_t)data->average_amplitude);
      auto out = std::make_shared<NetworkOpBurstParams>(
          (uint8_t*)buff, sizeof(uint32_t) * burst_length, 1);
      HOLOSCAN_LOG_INFO("Forwarding message to Network TX");
      op_output.emit(out, "burst_out");
    }
  }
  holoscan::Parameter<float> sample_rate;
  holoscan::Parameter<bool> to_screen;
  holoscan::Parameter<bool> to_tx;
};

// Calculates pulse descriptions.
class PulseDescriptorOp : public holoscan::Operator {
 public:
  HOLOSCAN_OPERATOR_FORWARD_ARGS(PulseDescriptorOp);
  PulseDescriptorOp() = default;

  void setup(holoscan::OperatorSpec& spec) override {
    spec.input<DetectedPulseSlice>("detected_pulses");
    spec.output<PulseDescription>("pulse_description");
  }

  void compute(holoscan::InputContext& op_input, holoscan::OutputContext& op_output,
               holoscan::ExecutionContext&) override {
    auto data = op_input.receive<DetectedPulseSlice>("detected_pulses");
    tensor_t<float, 0> sum_power{};
    tensor_t<float, 0> maximum_power{};
    // Compute statistics of of the pulse. Sum and average.
    sum(sum_power, data->power, 0);
    rmax(maximum_power, data->power, 0);
    hipStreamSynchronize(0);
    float average_amplitude = sum_power() / data->power.Size(0);
    auto out = std::make_shared<PulseDescription>(data->id,
                                                  data->offset,
                                                  data->offset + data->power.Size(0),
                                                  data->zero_bin,
                                                  maximum_power(),
                                                  sum_power(),
                                                  average_amplitude);
    op_output.emit(out, "pulse_description");
  }
};

// This is where the hard math is. This finds where in an array the rising and falling edges are i.e
// if your signal looks like this:
//        ______
// ______/      \_________
//       ^      ^
// 012345678901234567890
//
// This would forward a pulse between 6-13.
class ThresholdingOp : public holoscan::Operator {
 public:
  HOLOSCAN_OPERATOR_FORWARD_ARGS(ThresholdingOp);
  ThresholdingOp() = default;
  void setup(holoscan::OperatorSpec& spec) override {
    spec.input<TaggedSignalData>("fft_input");
    spec.output<DetectedPulseSlice>("detected_pulses");

    spec.param(threshold_param,
               "threshold",
               "Constant Threshold",
               "Samples greater than this threshold will trigger the detector",
               {});
    spec.param(max_pulse_count,
               "max_pulses",
               "Pulse Detection Limit",
               "The maximum number of pulses this detector can detect",
               {});
  }

  void compute(holoscan::InputContext& op_input, holoscan::OutputContext& op_output,
               holoscan::ExecutionContext&) override {
    auto data = op_input.receive<TaggedSignalData>("fft_input");
    auto thresh_mask = make_tensor<char>({data->signal.Size(0) + 1});
    tensor_t<float, 1> power = make_tensor<ftype>({data->signal.Size(0)});
    tensor_t<float, 0> threshold{};

    threshold() = threshold_param.get();

    (power = abs(data->signal)).run();
    // Prepend a zero so we can shift the data to test for rising and falling edges
    thresh_mask(0) = 0;
    (thresh_mask.Slice({1}, {data->signal.Size(0) + 1}) = (power > threshold)).run();

    tensor_t<int, 0> rising_edge_count{};
    tensor_t<int, 0> falling_edge_count{};

    auto rising_edges_index = make_tensor<unsigned int>({max_pulse_count.get()});
    auto falling_edges_index = make_tensor<unsigned int>({max_pulse_count.get()});

    auto right_shift = thresh_mask.Slice({0}, {data->signal.Size(0)});
    auto original = thresh_mask.Slice({1}, {data->signal.Size(0) + 1});

    // Threshold detector works by shifting a binary array by one and using logic to find the edges.
    // This:
    //        ______
    // ______/      \_________
    // Becomes This:
    // 00000011111111000000000
    //
    // And then gets shifted by 1
    // 00000011111111000000000
    // 00000001111111100000000
    //
    // As you can see the rising edge is when the top is when the original is
    // true and the shifted is false, while falling is reversed.
    auto rising_edge_op = original && !right_shift;
    auto falling_edge_op = !original && right_shift;

    find_idx(rising_edges_index, rising_edge_count, rising_edge_op, GT{0});
    find_idx(falling_edges_index, falling_edge_count, falling_edge_op, GT{0});

    hipStreamSynchronize(0);

    if (rising_edge_count() != falling_edge_count()) {
      HOLOSCAN_LOG_INFO("RISING AND FALLING EDGE COUNTS DO NOT MATCH");
      for (int i = 0; i < rising_edge_count(); i++) {
        std::cout << "Rising edge at " << rising_edges_index(i) << std::endl;
      }
      for (int i = 0; i < falling_edge_count(); i++) {
        std::cout << "Falling edge at " << falling_edges_index(i) << std::endl;
      }
    } else {
      for (int i = 0; i < rising_edge_count(); i++) {
        auto out = std::make_shared<DetectedPulseSlice>(
            data->id,
            rising_edges_index(i),
            // Falling edge is the index after the pulse since the slice is exclusive at the second
            // parameter this is correct.
            data->signal.Size(0) / 2,
            data->signal.Slice({rising_edges_index(i)}, {falling_edges_index(i)}),
            power.Slice({rising_edges_index(i)}, {falling_edges_index(i)}));
        op_output.emit(out, "detected_pulses");
      }
    }
  }

 private:
  holoscan::Parameter<int64_t> threshold_param;
  holoscan::Parameter<uint32_t> max_pulse_count;
};

// Takes an FFT and shift the FFT so that the center frequency is in the center.
class FFTOp : public holoscan::Operator {
 public:
  HOLOSCAN_OPERATOR_FORWARD_ARGS(FFTOp);
  FFTOp() = default;
  void setup(holoscan::OperatorSpec& spec) override {
    spec.input<TaggedSignalData>("signal_input");
    spec.output<TaggedSignalData>("fft_output");
  }

  void compute(holoscan::InputContext& op_input, holoscan::OutputContext& op_output,
               holoscan::ExecutionContext&) override {
    auto data = op_input.receive<TaggedSignalData>("signal_input");
    int size = data->signal.Size(0);
    auto shifted_fft = make_tensor<ComplexType, 1>({size});
    matx::fft(data->signal, data->signal);
    (data->signal = fftshift1D(data->signal)).run();
    op_output.emit(data, "fft_output");
  }
};

// Signal Generator Op that can generate a pure chirp. This may be swapped out with the network
// operator to generate signals without the use of an external network signal generator.
class SignalGeneratorOp : public holoscan::Operator {
 public:
  HOLOSCAN_OPERATOR_FORWARD_ARGS(SignalGeneratorOp);
  SignalGeneratorOp() = default;

  void setup(holoscan::OperatorSpec& spec) override {
    spec.output<TaggedSignalData>("signal_output");
    spec.param(number_samples,
               "samples_per_packet",
               "Samples per Packet",
               "The number of samples in a single packet in this generator.",
               {});
    spec.param(sample_rate, "sample_rate", "Samples per second", "Sample rate in Hz.", {});
    spec.param(chirp_starting_frequency,
               "chirp_starting_frequency",
               "Starting frequency",
               "Chirp Starting Frequency",
               {});
    spec.param(chirp_stop_frequency,
               "chirp_stopping_frequency",
               "Stopping frequency",
               "Chirp Stop Frequency",
               {});
  }

  void initialize() override {
    HOLOSCAN_LOG_INFO("SignalGeneratorOp::initialize()");
    holoscan::Operator::initialize();
    id = 0;
  }

  void compute(holoscan::InputContext& op_input, holoscan::OutputContext& op_output,
               holoscan::ExecutionContext&) override {
    double chirp_duration = (1.0 / sample_rate.get()) * number_samples.get();
    auto chirp_op = cchirp({number_samples.get()},
                           chirp_duration,
                           chirp_starting_frequency.get(),
                           chirp_duration,
                           chirp_stop_frequency.get());
    auto chirp = make_tensor<ComplexType>({number_samples.get()});
    auto out = std::make_shared<TaggedSignalData>(id, chirp);
    (out->signal = chirp_op).run();

    op_output.emit(out, "signal_output");
    id++;
  }

  int id;
  holoscan::Parameter<int32_t> number_samples;
  holoscan::Parameter<float> sample_rate;
  holoscan::Parameter<float> chirp_starting_frequency;
  holoscan::Parameter<float> chirp_stop_frequency;
};

// Processes network packets into the internal memory representation
class PacketToTensorOp : public holoscan::Operator {
 public:
  HOLOSCAN_OPERATOR_FORWARD_ARGS(PacketToTensorOp);
  PacketToTensorOp() = default;

  void setup(holoscan::OperatorSpec& spec) override {
    spec.input<NetworkOpBurstParams>("burst_in");
    spec.output<TaggedSignalData>("tensor_output");
  }

  void initialize() override {
    HOLOSCAN_LOG_INFO("Converting incoming packet data to Complex Tensor data");
    holoscan::Operator::initialize();
  }

  void compute(holoscan::InputContext& op_input, holoscan::OutputContext& op_output,
               holoscan::ExecutionContext&) override {
    auto packet = op_input.receive<NetworkOpBurstParams>("burst_in");
    int id = (packet->data[0] << 8) |
             packet->data[1];         // Getting the first 16 big-endian bits of the packet
    packet->data = packet->data + 2;  // Go up 16 bits to get to the actual packet data
    matx::index_t size =
        (packet->len - sizeof(int16_t)) / (sizeof(int16_t) * 2);  // Calc size in int16s
    auto nums = make_tensor<ComplexType>({size});
    int16_t* samples = (int16_t*)packet->data;
    for (int i = 0; i < size; i++) {
      nums(i) = ComplexType((int16_t)ntohs(samples[i * 2]),
                            (int16_t)ntohs(samples[i * 2 + 1]));  // fill tensor
    }
    auto out = std::make_shared<TaggedSignalData>(id, nums);
    op_output.emit(out, "tensor_output");
  }
};

class App : public holoscan::Application {
 public:
  void compose() override {
    using namespace holoscan;

    auto net_rx = make_operator<ops::BasicNetworkOpRx>("net_rx", from_config("network_rx"));
    auto convert = make_operator<PacketToTensorOp>("converter");
    auto fft = make_operator<FFTOp>("fft");
    auto thresh = make_operator<ThresholdingOp>("pulse_detector", from_config("pulse_detector"));
    auto descrip = make_operator<PulseDescriptorOp>("pulse_descriptor");
    auto printer = make_operator<PulsePrinterOp>("pulse_printer", from_config("printer"));
    auto net_tx = make_operator<ops::BasicNetworkOpTx>("net_tx", from_config("network_tx"));

    add_flow(net_rx, convert, {{"burst_out", "burst_in"}});
    add_flow(convert, fft, {{"tensor_output", "signal_input"}});
    add_flow(fft, thresh, {{"fft_output", "fft_input"}});
    add_flow(thresh, descrip, {{"detected_pulses", "detected_pulses"}});
    add_flow(descrip, printer, {{"pulse_description", "pulse_description"}});
    add_flow(printer, net_tx, {{"burst_out", "burst_in"}});
  }
};

int main(int argc, char** argv) {
  auto app = holoscan::make_application<App>();

  // Get the configuration
  auto config_path = std::filesystem::canonical(argv[0]).parent_path();
  config_path += "/simple_pdw_pipeline.yaml";
  app->config(config_path);

  app->run();

  return 0;
}
