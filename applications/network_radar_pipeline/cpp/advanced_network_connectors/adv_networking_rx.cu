#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "adv_networking_rx.h"  // TODO: Rename networking connectors

#if SPOOF_PACKET_DATA
/**
 * This function converts the packet count to packet metadata. We just treat the
 * packet count as if all of the packets are arriving in order and write the the
 * metadata accordingly. This functionalitycan be useful when testing, where we
 * have a packet generator that isn't generating packets that use our data format.
 */
__device__ __forceinline__ void gen_meta_from_pkt_cnt(RfMetaData* meta, const uint64_t pkt_cnt,
                                                      const uint16_t num_channels,
                                                      const uint16_t num_pulses,
                                                      const uint16_t num_samples,
                                                      const uint16_t pkts_per_pulse,
                                                      const uint16_t max_waveform_id) {
  const uint64_t pkts_per_channel = pkts_per_pulse * num_pulses;
  const uint64_t pkts_per_transmit = pkts_per_channel * num_channels;
  meta->waveform_id = static_cast<uint16_t>((pkt_cnt / pkts_per_transmit) % max_waveform_id);
  meta->channel_idx = static_cast<uint16_t>((pkt_cnt % pkts_per_transmit) / pkts_per_channel);
  meta->pulse_idx = static_cast<uint16_t>((pkt_cnt % pkts_per_channel) / pkts_per_pulse);
  meta->sample_idx = static_cast<uint16_t>(SPOOF_SAMPLES_PER_PKT * (pkt_cnt % pkts_per_pulse));
  meta->pkt_samples = min(num_samples - meta->sample_idx, SPOOF_SAMPLES_PER_PKT);
  meta->end_array = (pkt_cnt + 1) % pkts_per_transmit == 0;
}
#endif

__global__ void place_packet_data_kernel(complex_t* out, const void* const* const __restrict__ in,
                                         int* sample_cnt, bool* received_end,
                                         const size_t buffer_pos, const uint16_t pkt_len,
                                         const uint16_t buffer_size, const uint16_t num_channels,
                                         const uint16_t num_pulses, const uint16_t num_samples,
                                         const uint64_t total_pkts, const uint16_t pkts_per_pulse,
                                         const uint16_t max_waveform_id) {
  const uint32_t channel_stride = static_cast<uint32_t>(num_samples) * num_pulses;
  const uint32_t buffer_stride = num_channels * channel_stride;
  const uint32_t pkt_idx = blockIdx.x;

  // Warmup
  if (out == nullptr)
    return;
#if SPOOF_PACKET_DATA
  // Generate fake packet meta-data from the packet count
  RfMetaData meta_obj;
  RfMetaData* meta = &meta_obj;
  gen_meta_from_pkt_cnt(meta,
                        total_pkts + pkt_idx,
                        num_channels,
                        num_pulses,
                        num_samples,
                        pkts_per_pulse,
                        max_waveform_id);
  const complex_t* samples = reinterpret_cast<const complex_t*>(in[pkt_idx]);
#else
  const RfMetaData* meta = reinterpret_cast<const RfMetaData*>(in[pkt_idx]);
  const complex_t* samples = reinterpret_cast<const complex_t*>(in[pkt_idx]) + 2;
#endif

  // Make sure this isn't wrapping the buffer - drop if it is
  if (meta->waveform_id >= buffer_pos + buffer_size || meta->waveform_id < buffer_pos) { return; }

  const uint16_t buffer_idx = meta->waveform_id % buffer_size;
  if (meta->end_array && threadIdx.x == 0) { received_end[buffer_idx] = true; }

  // Compute pointer in buffer memory
  const uint32_t idx_offset = meta->sample_idx + meta->pulse_idx * num_samples +
                              meta->channel_idx * channel_stride + buffer_idx * buffer_stride;

  // Copy data
  for (uint16_t i = threadIdx.x; i < meta->pkt_samples; i += blockDim.x) {
    out[idx_offset + i] = samples[i];
  }

  if (threadIdx.x == 0) {
    // todo Smarter way than atomicAdd
    atomicAdd(&sample_cnt[buffer_idx], meta->pkt_samples);
  }
}

void place_packet_data(complex_t* out, const void* const* const in, int* sample_cnt,
                       bool* received_end, const size_t buffer_pos, const uint16_t pkt_len,
                       const uint32_t num_pkts, const uint16_t buffer_size,
                       const uint16_t num_channels, const uint16_t num_pulses,
                       const uint16_t num_samples, const uint64_t total_pkts,
                       const uint16_t pkts_per_pulse, const uint16_t max_waveform_id,
                       hipStream_t stream) {
  // Each thread processes an individual packet
  place_packet_data_kernel<<<num_pkts, 128, buffer_size * sizeof(int), stream>>>(out,
                                                                                 in,
                                                                                 sample_cnt,
                                                                                 received_end,
                                                                                 buffer_pos,
                                                                                 pkt_len,
                                                                                 buffer_size,
                                                                                 num_channels,
                                                                                 num_pulses,
                                                                                 num_samples,
                                                                                 total_pkts,
                                                                                 pkts_per_pulse,
                                                                                 max_waveform_id);
}

namespace holoscan::ops {

void AdvConnectorOpRx::setup(OperatorSpec& spec) {
  spec.input<std::shared_ptr<AdvNetBurstParams>>("burst_in");
  spec.output<std::shared_ptr<RFArray>>("rf_out");

  // Radar settings
  spec.param<uint16_t>(buffer_size_,
                       "buffer_size",
                       "Size of RF buffer",
                       "Max number of transmits that can be held at once",
                       {});
  spec.param<uint16_t>(
      num_channels_, "num_channels", "Number of channels", "Number of channels", {});
  spec.param<uint16_t>(
      num_pulses_, "num_pulses", "Number of pulses", "Number of pulses per channel", {});
  spec.param<uint16_t>(
      num_samples_, "num_samples", "Number of samples", "Number of samples per pulse", {});

  // Networking settings
  spec.param<bool>(split_boundary_,
                   "split_boundary",
                   "Header-data split boundary",
                   "Byte boundary where header and data is split",
                   true);
  spec.param<bool>(gpu_direct_,
                   "gpu_direct",
                   "GPUDirect enabled",
                   "GPUDirect is enabled for incoming packets",
                   true);
  spec.param<uint32_t>(batch_size_,
                       "batch_size",
                       "Batch size",
                       "Batch size in packets for each processing epoch",
                       1000);
  spec.param<uint16_t>(max_packet_size_,
                       "max_packet_size",
                       "Max packet size",
                       "Maximum packet size expected from sender",
                       9100);
}

void AdvConnectorOpRx::initialize() {
  HOLOSCAN_LOG_INFO("AdvConnectorOpRx::initialize()");
  holoscan::Operator::initialize();

  hipStreamCreateWithFlags(&proc_stream, hipStreamNonBlocking);

  // Assume all packets are the same size, specified in the config
  nom_payload_size_ = max_packet_size_.get() - PADDED_HDR_SIZE;

  // Total number of I/Q samples per array
  samples_per_arr = num_channels_.get() * num_pulses_.get() * num_samples_.get();

  // Configuration checks
  if (split_boundary_.get() && !gpu_direct_.get()) {
    HOLOSCAN_LOG_ERROR("If Header-Data Split mode is enabled, GPUDirect needs to be too");
    exit(1);
  }

  // Allocate memory and create CUDA streams for each concurrent batch
  for (int n = 0; n < num_concurrent; n++) {
    if (gpu_direct_.get()) {
      hipHostMalloc((void**)&h_dev_ptrs_[n], sizeof(void*) * batch_size_.get() * 2);
    }

    buffer_track = AdvBufferTracking(buffer_size_.get());
    make_tensor(rf_data,
                {buffer_size_.get(), num_channels_.get(), num_pulses_.get(), num_samples_.get()});

    hipStreamCreateWithFlags(&streams_[n], hipStreamNonBlocking);
    hipEventCreate(&events_[n]);
    // Warmup
    place_packet_data(nullptr,
                        nullptr,
                        0,
                        0,
                        0,
                        0,
                        16,
                        16,
                        0,
                        0,
                        0,
                        0,
                        0,
                        0,
                        streams_[n]);
    hipStreamSynchronize(streams_[n]);
  }

#if SPOOF_PACKET_DATA
  // Compute packets delivered per pulse and max waveform ID based on parameters
  const size_t spoof_pkt_size = sizeof(complex_t) * SPOOF_SAMPLES_PER_PKT + RFPacket::header_size();
  pkts_per_pulse = static_cast<uint16_t>(packets_per_pulse(spoof_pkt_size, num_samples_.get()));
  max_waveform_id =
      static_cast<uint16_t>(buffer_size_.get() * (65535 / buffer_size_.get()));  // Max of uint16_t
  HOLOSCAN_LOG_WARN("Spoofing packet metadata, ignoring packet header. Pkts / pulse: {}",
                    pkts_per_pulse);
  if (spoof_pkt_size >= max_packet_size_.get()) {
    HOLOSCAN_LOG_ERROR("Max packets size ({}) can't fit the expected samples ({})",
                       max_packet_size_.get(),
                       SPOOF_SAMPLES_PER_PKT);
    exit(1);
  }
#else
  // These are only used when spoofing packet metadata
  pkts_per_pulse = 0;
  max_waveform_id = 0;
#endif

  HOLOSCAN_LOG_INFO("AdvConnectorOpRx::initialize() complete");
}

std::vector<AdvConnectorOpRx::RxMsg> AdvConnectorOpRx::free_bufs() {
  std::vector<AdvConnectorOpRx::RxMsg> completed;

  // Loop over all batches, checking if any have completed
  while (out_q.size() > 0) {
    const auto first = out_q.front();
    if (hipEventQuery(first.evt) == hipSuccess) {
      completed.push_back(first);
      for (auto m = 0; m < first.num_batches; m++) {
        adv_net_free_all_pkts_and_burst(first.msg[m]);
      }
      out_q.pop();
    } else {
      break;
    }
  }
  return completed;
}

void AdvConnectorOpRx::free_bufs_and_emit_arrays(OutputContext& op_output) {
  std::vector<AdvConnectorOpRx::RxMsg> completed_msgs = free_bufs();
  if (completed_msgs.empty()) { return; }
  hipStream_t stream = completed_msgs[0].stream;

  buffer_track.transfer(hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);

  for (size_t i = 0; i < buffer_track.buffer_size; i++) {
    const size_t pos_wrap = (buffer_track.pos + i) % buffer_track.buffer_size;
    if (!buffer_track.received_end_h[pos_wrap]) { continue; }

    // Received End-of-Array (EOA) message, emit to downstream operators
    auto params =
        std::make_shared<RFArray>(rf_data.Slice<3>({static_cast<index_t>(pos_wrap), 0, 0, 0},
                                                   {matxDropDim, matxEnd, matxEnd, matxEnd}),
                                  0,
                                  proc_stream);

    op_output.emit(params, "rf_out");
    HOLOSCAN_LOG_INFO("Emitting {} with {}/{} samples",
                      buffer_track.pos + i,
                      buffer_track.sample_cnt_h[pos_wrap],
                      samples_per_arr);

    // Increment the tracker 'i' number of times. This allows us to not get hung on arrays
    // where the EOA was either dropped or missed. Ex: if the EOA for array 11 was dropped,
    // we will emit array 12 when its EOA arrives, incrementing from 10 -> 12.
    for (size_t j = 0; j <= i; j++) {
      buffer_track.increment();
      HOLOSCAN_LOG_INFO("Next waveform expected: {}", buffer_track.pos);
    }

    buffer_track.transfer(hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    break;
  }
}

void AdvConnectorOpRx::compute(InputContext& op_input, OutputContext& op_output,
                               ExecutionContext& context) {
  // todo Some sort of warm start for the processing stages?
  int64_t ttl_bytes_in_cur_batch_ = 0;

  auto burst_opt = op_input.receive<std::shared_ptr<AdvNetBurstParams>>("burst_in");
  if (!burst_opt) {
    free_bufs();
    return;
  }
  auto burst = burst_opt.value();

  // If packets are coming in from our non-GPUDirect queue, free them and move on
  if (adv_net_get_q_id(burst) == 0) {  // queue 0 is configured to be non-GPUDirect in yaml config
    adv_net_free_all_pkts_and_burst(burst);
    HOLOSCAN_LOG_INFO("Freeing CPU packets on queue 0");
    return;
  }

  // Header data split saves off the GPU pointers into a host-pinned buffer to reassemble later.
  // Once enough packets are aggregated, a reorder kernel is launched. In CPU-only mode the
  // entire burst buffer pointer is saved and freed once an entire batch is received.
  if (gpu_direct_.get()) {
    if (split_boundary_.get()) {
      for (int p = 0; p < adv_net_get_num_pkts(burst); p++) {
        h_dev_ptrs_[cur_idx][aggr_pkts_recv_ + p] = adv_net_get_seg_pkt_ptr(burst, 1, p);
        ttl_bytes_in_cur_batch_ +=
            adv_net_get_seg_pkt_len(burst, 0, p) + adv_net_get_seg_pkt_len(burst, 1, p);
      }
    } else {
      for (int p = 0; p < adv_net_get_num_pkts(burst); p++) {
        h_dev_ptrs_[cur_idx][aggr_pkts_recv_ + p] =
            reinterpret_cast<uint8_t*>(adv_net_get_pkt_ptr(burst, p)) + PADDED_HDR_SIZE;
        ttl_bytes_in_cur_batch_ += adv_net_get_burst_tot_byte(burst);
      }
    }
  }
  ttl_bytes_recv_ += ttl_bytes_in_cur_batch_;

  aggr_pkts_recv_ += adv_net_get_num_pkts(burst);

  HOLOSCAN_LOG_INFO("aggr_pkts_recv_ {} ttl_bytes_recv_ {} batch_size_ {}",
      aggr_pkts_recv_, ttl_bytes_recv_, batch_size_.get());

  // Once we've aggregated enough packets, do some work
  if (aggr_pkts_recv_ >= batch_size_.get()) {
    if (gpu_direct_.get()) {
      do {
        free_bufs_and_emit_arrays(op_output);
        if (out_q.size() == num_concurrent) {
          HOLOSCAN_LOG_ERROR("Fell behind in processing on GPU!");
          hipStreamSynchronize(streams_[cur_idx]);
        }
      } while (out_q.size() == num_concurrent);

      HOLOSCAN_LOG_DEBUG("place_packet_data cur_idx {}", cur_idx);
      // Copy packet I/Q contents to appropriate location in 'rf_data'
      place_packet_data(rf_data.Data(),
                        h_dev_ptrs_[cur_idx],
                        buffer_track.sample_cnt_d,
                        buffer_track.received_end_d,
                        buffer_track.pos,
                        nom_payload_size_,
                        aggr_pkts_recv_,
                        buffer_size_.get(),
                        num_channels_.get(),
                        num_pulses_.get(),
                        num_samples_.get(),
                        ttl_pkts_recv_,   // only needed if spoofing packets
                        pkts_per_pulse,   // only needed if spoofing packets
                        max_waveform_id,  // only needed if spoofing packets
                        streams_[cur_idx]);

      hipEventRecord(events_[cur_idx], streams_[cur_idx]);
      cur_msg_.stream = streams_[cur_idx];
      cur_msg_.evt = events_[cur_idx];
      out_q.push(cur_msg_);
      cur_msg_.num_batches = 0;

      ttl_pkts_recv_ += aggr_pkts_recv_;

      if (hipGetLastError() != hipSuccess) {
        HOLOSCAN_LOG_ERROR("CUDA error with {} packets in batch and {} bytes total",
                           batch_size_.get(),
                           batch_size_.get() * nom_payload_size_);
        exit(1);
      }
    } else {
      adv_net_free_all_pkts_and_burst(burst);
    }
    aggr_pkts_recv_ = 0;
    cur_idx = (++cur_idx % num_concurrent);
  }
}

void AdvConnectorOpRx::stop() {
  HOLOSCAN_LOG_INFO(
      "\n"
      "AdvConnectorOpRx exit report:\n"
      "--------------------------------\n"
      " - Processed bytes:     {}\n"
      " - Processed packets:   {}\n",
      ttl_bytes_recv_,
      ttl_pkts_recv_);
}

}  // namespace holoscan::ops
