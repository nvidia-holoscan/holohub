#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "adv_networking_tx.h"  // TODO: Rename networking connectors

namespace holoscan::ops {

void AdvConnectorOpTx::setup(OperatorSpec& spec) {
  spec.input<std::shared_ptr<RFChannel>>("rf_in");
  spec.output<std::shared_ptr<AdvNetBurstParams>>("burst_out");

  // Advanced network operator parameters
  spec.param<AdvNetConfigYaml>(cfg_,
    "cfg",
    "Configuration",
    "Configuration for the advanced network operator",
    AdvNetConfigYaml());

  // Radar parameters
  spec.param<uint16_t>(num_pulses_,
    "num_pulses",
    "Number of pulses",
    "Number of pulses per channel", {});
  spec.param<uint16_t>(num_channels_,
    "num_channels",
    "Number of channels",
    "Number of channels", {});
  spec.param<uint16_t>(waveform_length_,
    "waveform_length",
    "Waveform length",
    "Length of waveform", {});
  spec.param<uint16_t>(num_samples_,
    "num_samples",
    "Number of samples",
    "Number of samples per channel", {});

  // Networking parameters
  spec.param<uint16_t>(samples_per_packet_,
    "samples_per_packet",
    "Number of complex I/Q samples to send per packet",
    "Payload size computed based on samples/packet, not including <= L4 headers", 128);
  spec.param<uint16_t>(header_size_,
    "header_size",
    "Header size",
    "Header size on each packet from L4 and below", 42);
  spec.param<uint16_t>(udp_src_port_,
    "udp_src_port",
    "UDP source port",
    "UDP source port");
  spec.param<uint16_t>(udp_dst_port_,
    "udp_dst_port",
    "UDP destination port",
    "UDP destination port");
  spec.param<std::string>(ip_src_addr_,
    "ip_src_addr",
    "IP source address",
    "IP source address");
  spec.param<std::string>(ip_dst_addr_,
    "ip_dst_addr",
    "IP destination address",
    "IP destination address");
  spec.param<std::string>(eth_dst_addr_,
    "eth_dst_addr",
    "Ethernet destination address",
    "Ethernet destination address");
  spec.param<uint16_t>(port_id_,
    "port_id",
    "Interface number",
    "Interface number");
}

void AdvConnectorOpTx::initialize() {
  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::initialize()");
  register_converter<holoscan::ops::AdvNetConfigYaml>();
  holoscan::Operator::initialize();

  // Read some parameters from config
  if (cfg_.get().tx_.size() != 1 || cfg_.get().tx_[0].queues_.size() != 1) {
    HOLOSCAN_LOG_ERROR("Currently can only handle 1 Tx queue");
    return;
  }
  payload_size_ = RFPacket::packet_size(samples_per_packet_.get());
  batch_size_   = cfg_.get().tx_[0].queues_[0].common_.batch_size_;
  hds_          = cfg_.get().tx_[0].queues_[0].common_.hds_;
  gpu_direct_   = cfg_.get().tx_[0].queues_[0].common_.gpu_direct_;

  if (gpu_direct_ && hds_ == 0) {
    // TODO: GPU-only mode Eth+IP+UDP headers
    HOLOSCAN_LOG_ERROR("Not configured for GPU-only, GPUDirect requires HDS mode for now");
    return;
  }

  // Compute how many packets sent per array
  samples_per_pkt = (payload_size_ - RFPacket::header_size()) / sizeof(complex_t);
  pkt_per_pulse   = packets_per_pulse(payload_size_, num_samples_.get());
  num_packets_buf = packets_per_channel(payload_size_, num_pulses_.get(), num_samples_.get());
  HOLOSCAN_LOG_INFO("samples_per_pkt: {}", samples_per_pkt);
  HOLOSCAN_LOG_INFO("num_packets_buf: {}", num_packets_buf);

  if (num_packets_buf >= batch_size_) {
    // TODO: Figure out a better way to break up and send a large chunk of data
    HOLOSCAN_LOG_ERROR(
      "RF array size too large: [{}, {}] requires {} packets and the max batch size is set to {}",
      num_pulses_.get(), num_samples_.get(), num_packets_buf, batch_size_);
    return;
  }

  // Reserve memory
  buf_stride = RFPacket::packet_size(samples_per_pkt);
  buf_size   = num_packets_buf * buf_stride;
  if (!gpu_direct_) {
    // On CPU
    mem_buf_h_ = static_cast<uint8_t *>(malloc(buf_size));
    if (mem_buf_h_ == nullptr) {
      HOLOSCAN_LOG_ERROR("Failed to allocate {} bytes of CPU batch memory", buf_size);
      return;
    }
    packets_buf = new RFPacket[num_packets_buf];
    for (size_t i = 0; i < num_packets_buf; i++) {
      packets_buf[i] = RFPacket(&mem_buf_h_[i * buf_stride]);
    }
  } else {
    // On GPU
    for (int n = 0; n < num_concurrent; n++) {
      hipHostMalloc(&gpu_bufs[n], sizeof(uint8_t**) * batch_size_);
      hipStreamCreate(&streams_[n]);
      hipEventCreate(&events_[n]);
    }
    HOLOSCAN_LOG_INFO("Initialized {} streams and events", num_concurrent);
  }

  adv_net_format_eth_addr(eth_dst_, eth_dst_addr_.get());
  inet_pton(AF_INET, ip_src_addr_.get().c_str(), &ip_src_);
  inet_pton(AF_INET, ip_dst_addr_.get().c_str(), &ip_dst_);

  // ANO expects host order when setting
  ip_src_ = ntohl(ip_src_);
  ip_dst_ = ntohl(ip_dst_);

  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::initialize() complete");
}

/**
 * @brief Set UDP headers on Host memory
 */
AdvNetStatus AdvConnectorOpTx::set_cpu_hdr(AdvNetBurstParams *msg, const int pkt_idx) {
  AdvNetStatus ret;

  // Set Ethernet header
  if ((ret = adv_net_set_cpu_eth_hdr(msg,
                                     pkt_idx,
                                     eth_dst_)) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Failed to set Ethernet header for packet {}", pkt_idx);
    adv_net_free_all_burst_pkts_and_burst(msg);
    return ret;
  }

  // Remove Eth + IP size
  const auto ip_len = payload_size_ + header_size_.get() - (14 + 20);
  if ((ret = adv_net_set_cpu_ipv4_hdr(msg,
                                      pkt_idx,
                                      ip_len,
                                      17,
                                      ip_src_,
                                      ip_dst_)) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Failed to set IP header for packet {}", 0);
    adv_net_free_all_burst_pkts_and_burst(msg);
    return ret;
  }

  // Set UDP header
  if ((ret = adv_net_set_cpu_udp_hdr(msg,
                                     pkt_idx,
                                     // Remove Eth + IP + UDP headers
                                     payload_size_ + header_size_.get() - (14 + 20 + 8),
                                     udp_src_port_.get(),
                                     udp_dst_port_.get())) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Failed to set UDP header for packet {}", 0);
    adv_net_free_all_burst_pkts_and_burst(msg);
    return ret;
  }

  return AdvNetStatus::SUCCESS;
}

/**
 * @brief Copy I/Q RF data from tensor to packets
 */
__global__
void populate_packets_kernel(uint8_t **out_ptr,
                             complex_t *rf_data,
                             uint16_t waveform_id,
                             uint16_t channel_idx,
                             uint16_t samples_per_pkt,
                             uint16_t pkt_per_pulse,
                             uint16_t num_channels,
                             uint16_t num_samples,
                             uint16_t offset) {
  int pkt_idx   = blockIdx.x;
  int pulse_idx = blockIdx.y;
  int buf_idx   = pkt_idx + pulse_idx * pkt_per_pulse;
  uint8_t *pkt  = out_ptr[buf_idx] + offset;

  int sample_idx  = samples_per_pkt * pkt_idx;
  int pkt_samples = min(samples_per_pkt, num_samples - sample_idx);

  // Send EOA if this is the last packet of the array (i.e. the last packet of the
  // last pulse of the last channel)
  const bool last_channel = num_channels == (channel_idx + 1);
  const bool last_pulse   = gridDim.y == (pulse_idx + 1);  // gridDim.y == # pulses
  const bool last_packet  = gridDim.x == (pkt_idx + 1);  // gridDim.x == packets/pulse
  bool set_eoa = last_channel && last_pulse && last_packet;

  // Copy in meta data
  uint16_t *meta = reinterpret_cast<uint16_t *>(pkt);
  meta[0] = sample_idx;
  meta[1] = waveform_id;
  meta[2] = channel_idx;
  meta[3] = pulse_idx;
  meta[4] = pkt_samples;
  meta[5] = set_eoa;

  // Copy over payload
  auto out = reinterpret_cast<complex_t *>(pkt + RFPacket::payload_offset);
  auto in  = &rf_data[sample_idx + num_samples * pulse_idx];
  for (int samp = threadIdx.x; samp < pkt_samples; samp += blockDim.x) {
    out[samp] = in[samp];
  }
}

void AdvConnectorOpTx::populate_packets(uint8_t **out_ptr,
                                        complex_t *rf_data,
                                        uint16_t waveform_id,
                                        uint16_t channel_idx,
                                        uint16_t offset,
                                        hipStream_t stream) {
  const dim3 grid(pkt_per_pulse, num_pulses_.get(), 1);
  populate_packets_kernel<<<grid, 128, 0, stream>>>(
    out_ptr,
    rf_data,
    waveform_id,
    channel_idx,
    samples_per_pkt,
    pkt_per_pulse,
    num_channels_.get(),
    num_samples_.get(),
    offset);
}

void AdvConnectorOpTx::compute(InputContext& op_input,
                               OutputContext& op_output,
                               ExecutionContext& context) {
  // Check if GPU send is falling behind
  if (gpu_direct_ && (hipEventQuery(events_[cur_idx]) != hipSuccess)) {
    HOLOSCAN_LOG_ERROR("Falling behind on TX processing for index {}!", cur_idx);
    return;
  }

  // Input is pulse/sample data from a single channel
  auto rf_data = op_input.receive<std::shared_ptr<RFChannel>>("rf_in").value();
  if (rf_data == nullptr) {
    if (gpu_direct_ && out_q.size() > 0) {
      // If packet setup is done, send to ANO
      const auto first = out_q.front();
      if (hipEventQuery(first.evt) == hipSuccess) {
        // Transmit
        HOLOSCAN_LOG_INFO("AdvConnectorOpTx sending {} packets... ({}, {})",
                          adv_net_get_num_pkts(first.msg),
                          first.waveform_id,
                          first.channel_id);
        op_output.emit(first.msg, "burst_out");
        out_q.pop();
      }
    }
    return;
  }
  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::compute()");
  AdvNetStatus ret;

  /**
   * Spin waiting until a buffer is free. This can be stalled by sending
   * faster than the NIC can handle it. We expect the transmit operator to
   * operate much faster than the receiver since it's not having to do any
   * work to construct packets, and just copying from a buffer into memory.
  */
  auto msg = adv_net_create_burst_params();
  adv_net_set_hdr(msg, port_id_.get(), queue_id, num_packets_buf);

  while (!adv_net_tx_burst_available(msg)) {}
  if ((ret = adv_net_get_tx_pkt_burst(msg)) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Error returned from adv_net_get_tx_pkt_burst: {}",
      static_cast<int>(ret));
    return;
  }

  if (!gpu_direct_) {
    // Generate packets from RF data //todo Optimize this process
    index_t ix_buf = 0;
    index_t ix_max = static_cast<index_t>(num_samples_.get());
    for (index_t ix_pulse = 0; ix_pulse < num_pulses_.get(); ix_pulse++) {
      for (index_t ix_sample = 0; ix_sample < num_samples_.get(); ix_sample += samples_per_pkt) {
        // Slice to the samples this packet will send
        auto data = rf_data->data.Slice<1>(
          {ix_pulse, ix_sample},
          {matxDropDim, std::min(ix_sample + samples_per_pkt, ix_max)});

        // Use accessor functions to set payload
        packets_buf[ix_buf].set_waveform_id(rf_data->waveform_id);
        packets_buf[ix_buf].set_sample_idx(ix_sample);
        packets_buf[ix_buf].set_channel_idx(rf_data->channel_id);
        packets_buf[ix_buf].set_pulse_idx(ix_pulse);
        packets_buf[ix_buf].set_num_samples(data.Size(0));
        packets_buf[ix_buf].set_end_array(0);
        packets_buf[ix_buf].set_payload(data.Data(), rf_data->stream);
        ix_buf++;
      }
    }
    if (num_packets_buf != ix_buf || adv_net_get_num_pkts(msg) != ix_buf) {
      HOLOSCAN_LOG_ERROR("Not sending expected number of packets");
    }

    // Send end-of-array message if this is the last channel of the transmit
    const bool send_eoa_msg = rf_data->channel_id == (num_channels_.get() - 1);
    if (send_eoa_msg) {
      packets_buf[num_packets_buf - 1].set_end_array(1);
    }
  }

  // Setup packets
  int cpu_len;
  int gpu_len;
  for (int pkt_idx = 0; pkt_idx < adv_net_get_num_pkts(msg); pkt_idx++) {
    // For HDS mode or CPU mode populate the packet headers
    if (!gpu_direct_ || hds_ > 0) {
      ret = set_cpu_hdr(msg, pkt_idx);  // set packet headers
      if (ret != AdvNetStatus::SUCCESS) {
        return;
      }

      // Only set payload on CPU buffer if we're not using GPUDirect
      if (!gpu_direct_) {
        if ((ret = adv_net_set_cpu_udp_payload(msg,
                                               pkt_idx,
                                               packets_buf[pkt_idx].get_ptr(),
                                               payload_size_)) != AdvNetStatus::SUCCESS) {
          HOLOSCAN_LOG_ERROR("Failed to set UDP payload for packet {}", pkt_idx);
          adv_net_free_all_burst_pkts_and_burst(msg);
          return;
        }
      }
    }

    // Figure out the CPU and GPU length portions for ANO
    if (gpu_direct_ && hds_ > 0) {
      cpu_len = hds_;
      gpu_len = payload_size_;
    } else if (!gpu_direct_) {
      cpu_len = payload_size_ + header_size_.get();  // sizeof UDP header
      gpu_len = 0;
    } else {
      cpu_len = 0;
      gpu_len = payload_size_ + header_size_.get();  // sizeof UDP header
    }

    if (gpu_direct_) {
      gpu_bufs[cur_idx][pkt_idx] = reinterpret_cast<uint8_t *>(
        adv_net_get_gpu_pkt_ptr(msg, pkt_idx));
    }

    if ((ret = adv_net_set_pkt_len(msg, pkt_idx, cpu_len, gpu_len)) != AdvNetStatus::SUCCESS) {
      HOLOSCAN_LOG_ERROR("Failed to set lengths for packet {}", pkt_idx);
      adv_net_free_all_burst_pkts_and_burst(msg);
      return;
    }
  }

  // In GPU-only mode copy the header
  if (gpu_direct_ && hds_ == 0) {
    // TODO: GPU-only mode
  }

  // Populate packets with I/Q data
  if (gpu_direct_) {
    const auto offset = (hds_ > 0) ? 0 : header_size_.get();
    populate_packets(gpu_bufs[cur_idx],
                     rf_data->data.Data(),
                     rf_data->waveform_id,
                     rf_data->channel_id,
                     offset,
                     streams_[cur_idx]);
    hipEventRecord(events_[cur_idx], streams_[cur_idx]);
    out_q.push(TxMsg{msg, rf_data->waveform_id, rf_data->channel_id, events_[cur_idx]});
  }

  if (gpu_direct_) {
    // If packet setup is done, send to ANO
    const auto first = out_q.front();
    if (hipEventQuery(first.evt) == hipSuccess) {
      // Transmit
      HOLOSCAN_LOG_INFO("AdvConnectorOpTx sending {} packets... ({}, {})",
                        adv_net_get_num_pkts(first.msg),
                        first.waveform_id,
                        first.channel_id);
      op_output.emit(first.msg, "burst_out");
      out_q.pop();
    }
  } else {
    // Transmit
    HOLOSCAN_LOG_INFO("AdvConnectorOpTx sending {} packets... ({}, {})",
                      adv_net_get_num_pkts(msg),
                      rf_data->waveform_id,
                      rf_data->channel_id);
    op_output.emit(msg, "burst_out");
  }

  // Increment index
  cur_idx = (++cur_idx % num_concurrent);

  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::compute() done");
}

}  // namespace holoscan::ops
