#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <stdio.h>
#include "kernels.cuh"
#include "matx.h"

__global__ void populate_packets(uint8_t** gpu_bufs, uint16_t pkt_len, uint16_t offset) {
  int pkt = blockIdx.x;

  for (int samp = threadIdx.x; samp < pkt_len / 4; samp += blockDim.x) {
    auto p = reinterpret_cast<uint32_t*>(gpu_bufs[pkt] + offset + samp * sizeof(uint32_t));
    *p = (samp << 16) | (samp & 0xff);
  }
}

/**
 * @brief Populate each packet with a monotonically-increasing sequence
 *
 * @param gpu_bufs GPU packet pointer list from ANO "gpu_pkts"
 * @param pkt_len Length of each packet in bytes. Must be a multiple of 4
 * @param num_pkts Number of packets
 * @param offset Offset into packet to start
 * @param stream CUDA stream
 */
void populate_packets(uint8_t** gpu_bufs, uint16_t pkt_len, uint32_t num_pkts, uint16_t offset,
                      hipStream_t stream) {
  populate_packets<<<num_pkts, 256, 0, stream>>>(gpu_bufs, pkt_len, offset);
}

// Must be divisible by 4 bytes in this kernel!
__global__ void copy_headers(uint8_t** gpu_bufs, void* header, uint16_t hdr_size) {
  if (gpu_bufs == nullptr) return;

  int pkt = blockIdx.x;

  for (int samp = threadIdx.x; samp < hdr_size / 4; samp += blockDim.x) {
    auto p = reinterpret_cast<uint32_t*>(gpu_bufs[pkt] + samp * sizeof(uint32_t));
    *p = *(reinterpret_cast<uint32_t*>(header) + samp);
  }
}

__global__ void print_packets(uint8_t** gpu_bufs) {
  uint8_t* p = gpu_bufs[0];
  for (int i = 0; i < 64; i++) { printf("%02X ", p[i]); }

  printf("\n");
}

void copy_headers(uint8_t** gpu_bufs, void* header, uint16_t hdr_size, uint32_t num_pkts,
                  hipStream_t stream) {
  copy_headers<<<num_pkts, 32, 0, stream>>>(gpu_bufs, header, hdr_size);
}
